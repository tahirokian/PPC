#include "hip/hip_runtime.h"
#include <numeric>     // std::accumulate
#include <algorithm>   // std::transform, std::copy
#include <vector>      // std::vector
#include <cmath>       // std::sqrt, std::ceil
#include "cp.h"
#include <hip/hip_runtime.h>

#define TILE 16

//Kernel code
__global__ void correlationKernel(float* input, /*float* inputTr,*/ float* output, int nx, int ny){
  int bx = blockDim.x, by = blockDim.y;
  int tx = threadIdx.x, ty = threadIdx.y;
  int y = ty + blockIdx.y * TILE;
  int x = tx + blockIdx.x * TILE;
  __shared__ float subInX[TILE][TILE];
  __shared__ float subInY[TILE][TILE];
  //if (x >= ny || y >= ny) // Exit if outside, do not calculate lower triangle
  //  return;
  float sum = 0.0;
  #pragma unroll
  for (int i = 0; i < (TILE + nx -1)/TILE; ++i){	//rounding up --> number of blocks
    if ((i * TILE + tx) < nx && y < ny)			//checking thread < nx and selected row < ny
      subInX[ty][tx] = input[y*nx + i*TILE + tx];
    else
      subInX[ty][tx] = 0;
    if ((i * TILE + ty) < nx && x < ny)			//checking thread < nx and selected row < ny
      subInY[ty][tx] = input[x*nx + i*TILE + ty];
    else
      subInY[ty][tx] = 0;
    __syncthreads();
    for (int j = 0; j < TILE; ++j)
      sum += subInX[ty][j]*subInY[j][tx];
    __syncthreads();
  }
  if (y < ny && x < ny)
    output[((blockIdx.y*by+ty)*ny)+(blockIdx.x*bx)+tx] = sum;
}

void correlate(int ny, int nx, const float* data, float* result) {
  float rowMean, normFactor;
  int rowStart, rowEnd;
  size_t inputSize = ny * nx;
  size_t outputSize = ny * ny;
  float* hostIn = 0;
  //float* hostInTr = 0;
  float* deviceIn = 0;
  float* deviceInTr = 0;
  float* deviceOut = 0;
  std::vector<float> zeroMeanVec(nx), elemSqrdVec(nx);
  hipHostMalloc((void**) &hostIn, inputSize * sizeof(float));
  //hipHostMalloc((void**) &hostInTr, inputSize * sizeof(float));
  hipMalloc((void**) &deviceIn, inputSize * sizeof(float));
  //hipMalloc((void**) &deviceInTr, inputSize * sizeof(float));
  hipMalloc((void**) &deviceOut, outputSize * sizeof(float));
  dim3 blockSize(TILE,TILE);                                                          //block of 8x8
  dim3 gridSize(std::ceil(float(ny)/blockSize.x), std::ceil(float(ny)/blockSize.y));  //grid of (ny/8)x(ny/8)
  for(int y = 0; y < ny; ++y){
    rowStart = y*nx;
    rowEnd = nx+rowStart;
    //Find mean of the current row
    rowMean = std::accumulate(data+rowStart, data+rowEnd, 0.0) / float(nx);
    //Subtract each element of the current row from mean to make row zero mean
    std::transform(data+rowStart, data+rowEnd, zeroMeanVec.begin(), [&rowMean](float val){ return (val - rowMean);});
    //Find square of each element of the current row
    std::transform(zeroMeanVec.begin(), zeroMeanVec.end(), elemSqrdVec.begin(), [](float val){ return (val * val);});
    //Find normalization factor  of the current row
    normFactor = std::sqrt(std::accumulate(elemSqrdVec.begin(), elemSqrdVec.end(), 0.0));
    //Normalize the current row so that the sum of the squares of the elements of the row is 1 with zero mean
    std::transform(zeroMeanVec.begin(), zeroMeanVec.end(), zeroMeanVec.begin(), [&normFactor](float val){ return (val / normFactor);});
    //Save the normalized result in a matrix of dimension ny*nx
    std::copy(zeroMeanVec.begin(), zeroMeanVec.end(), hostIn+rowStart);
  }
  //Matrix transpose
  //for (int j=0; j<ny; ++j){
  //  for (int i=0; i<nx; ++i){
  //    hostInTr[j*nx+i] = hostIn[j*nx+i];
  //  }
  //}

  //Copy host data to GPU
  hipMemcpy(deviceIn, hostIn, inputSize * sizeof(float), hipMemcpyHostToDevice);
  //Copy hostTranspose data to GPU
  //hipMemcpy(deviceInTr, hostInTr, inputSize * sizeof(float), hipMemcpyHostToDevice);
  //Kernel call
  correlationKernel<<<gridSize, blockSize>>>(deviceIn, /*deviceInTr,*/ deviceOut, nx, ny);
  //Copy GPU data to host
  hipMemcpy(result, deviceOut, outputSize * sizeof(float), hipMemcpyDeviceToHost);
  //Free memory
  hipHostFree(hostIn);
  //hipHostFree(hostInTr);
  hipFree(deviceIn);
  hipFree(deviceInTr);
  hipFree(deviceOut);
}
